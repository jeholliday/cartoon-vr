#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <opencv2/opencv.hpp>

using namespace cv;

__device__ float
squared_l3_distance(float x_1, float y_1, float z_1, float x_2, float y_2, float z_2) {
  return (x_1 - x_2) * (x_1 - x_2)
       + (y_1 - y_2) * (y_1 - y_2)
       + (z_1 - z_2) * (z_1 - z_2);
}

__global__ void assign_clusters(const cv::cuda::PtrStepSzf data,
                                int row_size,
                                cv::cuda::PtrStepSzf means,
                                cv::cuda::PtrStepSzf new_sums,
                                int k,
                                cv::cuda::PtrStepSz<int32_t> counts) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= row_size) return;

  // Make global loads once.
  const float x = data(0, index);
  const float y = data(1, index);
  const float z = data(2, index);

  float best_distance = FLT_MAX;
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance = squared_l3_distance(x, y, z,
        means(0, cluster), means(1, cluster), means(2, cluster));
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
    }
  }

  atomicAdd(&new_sums(0, best_cluster), x);
  atomicAdd(&new_sums(1, best_cluster), y);
  atomicAdd(&new_sums(2, best_cluster), z);
  atomicAdd(&counts(0, best_cluster), 1);
}

__global__ void compute_new_means_and_reset(cv::cuda::PtrStepSzf means,
                                            cv::cuda::PtrStepSzf new_sums,
                                            cv::cuda::PtrStepSz<int32_t> counts) {
  const int cluster = threadIdx.x;
  const int count = max(1, counts[cluster]);
  means(0, cluster) = new_sums(0, cluster) / count;
  means(1, cluster) = new_sums(1, cluster) / count;
  means(2, cluster) = new_sums(2, cluster) / count;

  new_sums(0, cluster) = 0;
  new_sums(1, cluster) = 0;
  new_sums(2, cluster) = 0;
  counts(0, cluster) = 0;
}

static Mat generate_random_means(Mat src, size_t k){
  Mat img = src.reshape(3, 1);
  img.convertTo(img, CV_32FC3);

  std::mt19937 rng(std::random_device{}());
  std::uniform_int_distribution<int> distribution(0,img.cols);

  Mat centers(k, 1, CV_32FC3);
  for(int i = 0; i < k; ++i){
    centers.at<Vec3f>(i, 0) = img.at<Vec3f>(0, distribution(rng));
  }
  centers = centers.reshape(1, k);
  return centers.t();
}

static cv::cuda::GpuMat g_data, g_means, g_sums, g_counts;

Mat kmeans(Mat src, Mat means, size_t k, size_t max_iterations){
  Mat data;
  resize(src, data, src.size()/2); // Down-sample original image
  data = data.reshape(1, data.total());
  data = data.t();
  data.convertTo(data, CV_32F);
  g_data.upload(data);

  if(means.empty()){
    means = generate_random_means(src, k);
  }
  g_means.upload(means);

  if(g_sums.empty() || g_sums.size() != means.size()) {
    g_sums.create(means.size(), CV_32F);
  }
  g_sums.setTo(Scalar::all(0.0));

  if(g_counts.empty() || g_counts.size() != means.size()) {
    g_counts.create(means.size(), CV_32S);
  }
  g_counts.setTo(Scalar::all(0.0));

  const size_t threads = 1024;
  size_t number_of_elements = g_data.cols;
  int blocks = (number_of_elements + threads - 1) / threads;

  for (size_t iteration = 0; iteration < max_iterations; ++iteration) {
    assign_clusters <<<blocks, threads>>>(g_data, number_of_elements, g_means, g_sums, k, g_counts);
    hipDeviceSynchronize();
    compute_new_means_and_reset<<<1, k>>>(g_means, g_sums, g_counts);
    hipDeviceSynchronize();

    Mat new_means;
    g_means.download(new_means);
    if(norm(means, new_means) < 1.0){
      // Stop early if less than threshold
      break;
    }
    means = new_means;
  }
  return means;
}